
#include "hip/hip_runtime.h"

#include <iostream>

#include <stdio.h>
#include <math.h>
#include <chrono>

bool check_parity(unsigned long long *number) {
	return *number & 1;	//oszcz�dzone 3 instrukcje w por�wnaniu do modulo 2
}

bool isPrimaryNumber(unsigned long long *number) {

	bool isPrime = true;

	if (!check_parity(number))
		return false;
	else {
		unsigned long squared_number = sqrt(*number);

		for (unsigned long divider = 3; divider <= squared_number; divider += 2) {
			if (!(*number % divider)) {
				isPrime = false;
				break;
			}
		}
	}
	return isPrime;
}

__global__ void isPrime(bool *is_prime, unsigned long long *number, double max_divider) {

	unsigned long divider = (threadIdx.x + blockIdx.x * blockDim.x) * 2 + 3;

	if (divider <= max_divider) {
		if (!(*number % divider)) {
			*is_prime = false;
		}
	}
}

int main()
{
	unsigned long long number;
	unsigned long long *p_number = &number;

	std::cout << "Podaj liczbe do sprawdzenia: ";
	std::cin >> number;
	
	std::chrono::steady_clock::time_point beginCPU = std::chrono::steady_clock::now();
	bool is_prime = isPrimaryNumber(p_number);
	std::chrono::steady_clock::time_point endCPU = std::chrono::steady_clock::now();
	int duration = std::chrono::duration_cast<std::chrono::milliseconds>(endCPU - beginCPU).count();

	if (is_prime) {
		std::cout << "Sprawdzana liczba " << number << ", jest liczba pierwsza. \nUplynelo " <<
			duration << " milisekund.";
	}
	else {
		std::cout << "Sprawdzana liczba " << number << ", nie jest liczba pierwsza. \nUplynelo " <<
			duration << " milisekund.";
	}
	std::chrono::steady_clock::time_point beginGPU = std::chrono::steady_clock::now();

	////////////////////////////////////////////////////////////////////////////////////////////////////////////
	bool is_primeGPU = true;
	bool *p_is_prime = &is_primeGPU;

	double max_divider = sqrt(number);	

	unsigned long total_threads = max_divider / 2;  //check for errors?
	const int threads_per_block = 128;
	
	int blocks_number = (total_threads + threads_per_block - 1) / threads_per_block;


	unsigned long long *d_number;
	int size = sizeof(unsigned long long);

	bool *d_is_prime;

	hipMalloc((void**)&d_number, size);
	hipMalloc((void**)&d_is_prime, sizeof(bool));

	hipMemcpy(d_number, p_number, size, hipMemcpyHostToDevice);
	hipMemcpy(d_is_prime, p_is_prime, sizeof(bool), hipMemcpyHostToDevice);

	if (!check_parity(p_number)) {
		is_primeGPU = false;
	}
	else {
		isPrime <<<blocks_number, threads_per_block >>> (d_is_prime, d_number, max_divider);
	}
	hipMemcpy(p_is_prime, d_is_prime, sizeof(bool), hipMemcpyDeviceToHost);

	std::chrono::steady_clock::time_point endGPU = std::chrono::steady_clock::now();
	int durationGPU = std::chrono::duration_cast<std::chrono::milliseconds>(endGPU - beginGPU).count();
	if (is_primeGPU) {
		std::cout << "Sprawdzana liczba " << number << ", jest liczba pierwsza. \nUplynelo " <<
			durationGPU << " milisekund.";
	}
	else {
		std::cout << "Sprawdzana liczba " << number << ", nie jest liczba pierwsza. \nUplynelo " <<
			durationGPU << " milisekund.";
	}

    return 0;
}
